
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <algorithm>
using namespace std;
 
int a[35];
 
//【方法1】排序後取中間那一個。
int median(int i){
    int b[3] = {a[i-1], a[i-2], a[i-3]};
    sort(b, b+3);
    return b[1];
}
 
int main() {
    ios_base::sync_with_stdio(false);
    cin.tie(0);
    int n;
    while (cin >> n){
        for (int i=0; i<n; i++){
            cin >> a[i];
        }
        for (int i=3; i<n; i++){
            if (abs(a[i] - a[i-1]) < 5) continue;
            a[i] = median(i);
        }
        for (int i=0; i<n; i++)
            cout << a[i] << ' ';
        cout << '\n';
    }
 
    return 0;
}
